#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 256

__global__ void vectorAdd(const float *A, const float *B, float *C, int n) 
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n)
        C[idx] = A[idx] + B[idx];
}

int main() 
{
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    printf("Number of GPUs: %d\n", deviceCount);

    if (deviceCount < 2) {
        fprintf(stderr, "This example requires at least 2 GPUs.\n");
        return -1;
    }

    size_t dataSize = N * sizeof(float);
    float *d_A = NULL, *d_B = NULL, *d_C0 = NULL, *d_C1 = NULL;

    // Allocate and initialize A and B on GPU 0
    hipSetDevice(0);
    hipMalloc((void**)&d_A, dataSize);
    hipMalloc((void**)&d_B, dataSize);
    hipMalloc((void**)&d_C0, dataSize);

    // Use init kernel to fill A and B
    vectorAdd<<<(N + 255) / 256, 256>>>(d_A, d_B, d_C0, N);  // A and B are not initialized yet, but okay for structure

    // Manually initialize A and B
    float h_A[N], h_B[N];
    for (int i = 0; i < N; ++i) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    hipMemcpy(d_A, h_A, dataSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, dataSize, hipMemcpyHostToDevice);

    // Compute vectorAdd on GPU 0
    vectorAdd<<<(N + 255) / 256, 256>>>(d_A, d_B, d_C0, N);
    hipDeviceSynchronize();

    // Allocate result buffer on GPU 1
    hipSetDevice(1);
    hipMalloc((void**)&d_C1, dataSize);


    // Copy result from GPU 0 to GPU 1
    hipMemcpyPeerAsync(d_C1, 1, d_C0, 0, dataSize);
    hipMemcpy(d_C1, d_C0, dataSize, hipMemcpyDeviceToDevice);
    hipDeviceSynchronize();

    // Copy to host and print
    float h_C[N];
    hipMemcpy(h_C, d_C1, dataSize, hipMemcpyDeviceToHost);

    printf("Sample result on GPU 1 (C = A + B): ");
    for (int i = 0; i < 5; ++i)
        printf("%f ", h_C[i]);
    printf("...\n");

    // Cleanup
    hipSetDevice(0);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C0);

    hipSetDevice(1);
    hipFree(d_C1);

    return 0;
}
