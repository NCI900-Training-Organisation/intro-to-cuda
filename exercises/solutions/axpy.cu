
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
  Helper functions for error checking
*/
#define cuda_check_impl(cmd, abort) {                  \
  hipError_t status = (cmd);                          \
  if (status != hipSuccess) {                         \
    fprintf(stderr, "CUDA Error: %s (%s:%d)\n",        \
      hipGetErrorString(status), __FILE__, __LINE__); \
    if (abort) exit(status);                           \
  }                                                    \
}
#define cuda_check(cmd) cuda_check_impl((cmd), 1)
#define cuda_check_noabort(cmd) cuda_check_impl((cmd), 0)


/*
  Helper functions for calculating grid size
*/
#define NBLOCKS(N, BLOCK_SIZE) (((N) + (BLOCK_SIZE) - 1)/(BLOCK_SIZE))
#define DEF_BLOCK 256
#define DEF_GRID(N) NBLOCKS((N), DEF_BLOCK)


/*
 TODO: Add your axpy function here
*/
__global__ void axpy(const float a, const float* X, const float* Y, float* Z, const int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) Z[idx] = a * X[idx] + Y[idx];
}


/*
  Test the axpy function to make sure it gives correct results
*/
typedef enum {SUCCESS=0, FAIL} TestResult;

TestResult test_axpy(const int N) {
  // Allocate memory
  float* X = (float*)malloc(sizeof(*X) * N);
  float* Y = (float*)malloc(sizeof(*Y) * N);
  float* Z = (float*)malloc(sizeof(*Z) * N);
  
  // Set up initial values of a, X and Y
  float a = 2.0;
  for (int i = 0; i < N; ++i) {
    X[i] = i;
    Y[i] = 2.*(N-i-1);
  }
  
  // TODO: allocate memory on the GPU for X, Y, Z
  float* X_d; cuda_check(hipMalloc((void**)&X_d, sizeof(*X_d) * N));
  float* Y_d; cuda_check(hipMalloc((void**)&Y_d, sizeof(*Y_d) * N));
  float* Z_d; cuda_check(hipMalloc((void**)&Z_d, sizeof(*Z_d) * N));
  
  // TODO: Copy X and Y to the GPU
  cuda_check(hipMemcpy(X_d, X, sizeof(*X)*N, hipMemcpyHostToDevice));
  cuda_check(hipMemcpy(Y_d, Y, sizeof(*Y)*N, hipMemcpyHostToDevice));
  
  // TODO: Call the axpy kernel with at least N threads
  axpy<<<DEF_GRID(N), DEF_BLOCK>>>(a, X_d, Y_d, Z_d, N);
  cuda_check(hipGetLastError());
  
  // TODO: Copy the result back into Z
  cuda_check(hipMemcpy(Z, Z_d, sizeof(*Z)*N, hipMemcpyDeviceToHost));
  
  // Check the results are correct
  for (int i = 0; i < N; ++i) {
    if (Z[i] != a*X[i] + Y[i]) return FAIL;
  }
  return SUCCESS;
}


int main(void) {
  int TESTS[] = {1024, 10000, 500000};
  for (int i = 0; i < sizeof(TESTS)/sizeof(*TESTS); ++i) {
    printf("Testing axpy for N = %-10i...  ", TESTS[i]);
    fflush(stdout);
    if (test_axpy(TESTS[i]) == SUCCESS)
      printf("Passed!\n");
    else {
      printf("Failed!\n");
      return 1;
    }
  }
  printf("\nAll tests passed!\n");
  return 0;
}