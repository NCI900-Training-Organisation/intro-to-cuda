#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

/**
 * Macro to check CUDA runtime errors
 */
#define CHECK_CUDA(call)                                                      \
    do {                                                                      \
        hipError_t err = call;                                               \
        if (err != hipSuccess) {                                             \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",                      \
                    __FILE__, __LINE__, hipGetErrorString(err));             \
            exit(EXIT_FAILURE);                                               \
        }                                                                     \
    } while (0)

/**
 * Map cuBLAS status codes to human-readable strings
 */
const char* cublasGetErrorString(hipblasStatus_t status) {
    switch (status) {
        case HIPBLAS_STATUS_SUCCESS: return "CUBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "CUBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "CUBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "CUBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "CUBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR: return "CUBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "CUBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "CUBLAS_STATUS_INTERNAL_ERROR";
        default: return "Unknown cuBLAS error";
    }
}

/**
 * Macro to check cuBLAS errors
 */
#define CHECK_CUBLAS(call)                                                    \
    do {                                                                      \
        hipblasStatus_t status = call;                                         \
        if (status != HIPBLAS_STATUS_SUCCESS) {                                \
            fprintf(stderr, "cuBLAS error at %s:%d: %s\n",                    \
                    __FILE__, __LINE__, cublasGetErrorString(status));        \
            exit(EXIT_FAILURE);                                               \
        }                                                                     \
    } while (0)

int main(void)
{
    int N = 1000;        // Matrix dimension (N x N)
    int runs = 5;        // Number of GEMM runs

    // -------------------------------
    // 1. Create cuBLAS handle
    // -------------------------------
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // -------------------------------
    // 2. Set GPU device
    // -------------------------------
    int32_t gpuId = 0; 
    CHECK_CUDA(hipSetDevice(gpuId));

    // -------------------------------
    // 3. Create CUDA stream
    // -------------------------------
    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));
    CHECK_CUBLAS(hipblasSetStream(handle, stream));

    // -------------------------------
    // 4. Allocate device memory
    // -------------------------------

     // Allocate matrices on CPU
    double *hostPtrA = (double *)malloc(sizeof(double) * N * N);
    double *hostPtrB = (double *)malloc(sizeof(double) * N * N);
    double *hostPtrC = (double *)malloc(sizeof(double) * N * N);

    double *devPtrA, *devPtrB, *devPtrC;
    CHECK_CUDA(hipMalloc((double **)&devPtrA, sizeof(double) * N * N));
    CHECK_CUDA(hipMalloc((double **)&devPtrB, sizeof(double) * N * N));
    CHECK_CUDA(hipMalloc((double **)&devPtrC, sizeof(double) * N * N));

    // Initialize matrices with some values
    for (int i = 0; i < N * N; i++) {
        hostPtrA[i] = 1.0;
        hostPtrB[i] = 2.0;
        hostPtrC[i] = 0.0;
    }

    CHECK_CUDA(hipMemcpy(devPtrA, hostPtrA, sizeof(double) * N * N, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(devPtrB, hostPtrB, sizeof(double) * N * N, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(devPtrC, hostPtrC, sizeof(double) * N * N, hipMemcpyHostToDevice));

    // -------------------------------
    // 5. Set GEMM scalars
    // -------------------------------
    double alpha = 1.0;
    double beta  = 1.0;

    // -------------------------------
    // 6. Create CUDA events for timing
    // -------------------------------
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // -------------------------------
    // 7. Launch GEMM 'runs' times
    // -------------------------------
    for (int i = 0; i < runs; i++) {
        // Record start event
        CHECK_CUDA(hipEventRecord(start, stream));

        // Perform matrix multiplication: C = alpha * A * B + beta * C
        CHECK_CUBLAS(hipblasDgemm(handle, 
            HIPBLAS_OP_N, 
            HIPBLAS_OP_N, 
            N, 
            N, 
            N, 
            &alpha,
            devPtrA, N,
            devPtrB, N,
            &beta,
            devPtrC, N));

        // Record stop event and synchronize
        CHECK_CUDA(hipEventRecord(stop, stream));
        CHECK_CUDA(hipEventSynchronize(stop));

        // Compute elapsed time in milliseconds
        float elapsed_ms = 0;
        CHECK_CUDA(hipEventElapsedTime(&elapsed_ms, start, stop));

        printf("Run %d completed in %.3f ms.\n", i + 1, elapsed_ms);
    }

    // -------------------------------
    // 8. Cleanup
    // -------------------------------
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    CHECK_CUDA(hipFree(devPtrA));
    CHECK_CUDA(hipFree(devPtrB));
    CHECK_CUDA(hipFree(devPtrC));
    CHECK_CUDA(hipStreamDestroy(stream));
    CHECK_CUBLAS(hipblasDestroy(handle));

    return 0;
}
