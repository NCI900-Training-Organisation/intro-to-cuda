#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

#define NUM_STREAMS 4
#define NUM_ITERATIONS 10

__global__ void vectorAdd2D(const float *A, const float *B, float *C, int N) 
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * gridDim.x * blockDim.x + col;

    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() 
{
    const int N = 1 << 20; // 1 million elements per vector
    size_t size = N * sizeof(float);

    float *A[NUM_ITERATIONS], *B[NUM_ITERATIONS], *C[NUM_ITERATIONS];
    hipStream_t streams[NUM_STREAMS];

    // Create streams
    for (int s = 0; s < NUM_STREAMS; ++s) {
        hipStreamCreate(&streams[s]);
    }

    // Allocate unified memory
    for (int i = 0; i < NUM_ITERATIONS; ++i) {
        hipMallocManaged(&A[i], size);
        hipMallocManaged(&B[i], size);
        hipMallocManaged(&C[i], size);
    }

    // Configure block and grid
    dim3 blockDim(16, 16);
    int totalThreads = blockDim.x * blockDim.y;
    int gridSize = (N + totalThreads - 1) / totalThreads;
    dim3 gridDim((int)ceil(sqrt((float)gridSize)), (int)ceil(sqrt((float)gridSize)));

    // Create overall timing events
    hipEvent_t overallStart, overallStop;
    hipEventCreate(&overallStart);
    hipEventCreate(&overallStop);

    // Start overall timer
    hipEventRecord(overallStart);

    // Launch kernels asynchronously
    for (int iter = 0; iter < NUM_ITERATIONS; ++iter) {
        int stream_id = iter % NUM_STREAMS;

        // Initialize inputs
        for (int i = 0; i < N; ++i) {
            A[iter][i] = float(iter);
            B[iter][i] = float(i % 100);
        }

        vectorAdd2D<<<gridDim, blockDim, 0, streams[stream_id]>>>(
            A[iter], B[iter], C[iter], N);

    }

    // Wait for all streams to finish
    for (int s = 0; s < NUM_STREAMS; ++s) {
        hipStreamSynchronize(streams[s]);
    }

    // Stop overall timer
    hipEventRecord(overallStop);
    hipEventSynchronize(overallStop);

    float totalTime = 0;
    hipEventElapsedTime(&totalTime, overallStart, overallStop);
    printf("Total time for %d iterations using %d streams: %.4f ms\n", NUM_ITERATIONS, NUM_STREAMS, totalTime);

    // Verify results
    bool allPassed = true;
    for (int iter = 0; iter < NUM_ITERATIONS; ++iter) {
        for (int i = 0; i < N; ++i) {
            float expected = float(iter) + float(i % 100);
            if (C[iter][i] != expected) {
                printf("Mismatch at iter %d, index %d: got %.1f, expected %.1f\n",
                       iter, i, C[iter][i], expected);
                allPassed = false;
                break;
            }
        }
    }

    printf("Result verification: %s\n", allPassed ? "PASSED" : "FAILED");

    // Cleanup
    for (int i = 0; i < NUM_ITERATIONS; ++i) {
        hipFree(A[i]);
        hipFree(B[i]);
        hipFree(C[i]);
    }

    for (int s = 0; s < NUM_STREAMS; ++s) {
        hipStreamDestroy(streams[s]);
    }

    hipEventDestroy(overallStart);
    hipEventDestroy(overallStop);

    return 0;
}
