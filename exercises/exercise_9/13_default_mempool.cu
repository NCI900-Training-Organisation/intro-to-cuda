#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel: simple vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int N) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

void printPoolUsage(hipMemPool_t pool, const char* label) {
    size_t current, high;
    hipMemPoolGetAttribute(pool, hipMemPoolAttrReservedMemCurrent, &current);
    hipMemPoolGetAttribute(pool, hipMemPoolAttrReservedMemHigh, &high);

    printf("%s:\n", label);
    printf("  Reserved memory:     %lu bytes (%.2f MB)\n", current, current / (1024.0 * 1024));
    printf("  Peak reserved memory: %lu bytes (%.2f MB)\n\n", high, high / (1024.0 * 1024));
}

int main() 
{
    const int N = 1 << 16;
    const int size = N * sizeof(float);

    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize host input arrays
    for (int i = 0; i < N; ++i) {
        h_A[i] = i;
        h_B[i] = 2 * i;
    }

    float *d_A, *d_B, *d_C;
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Get the default memory pool for device 0
    hipMemPool_t defaultPool;
    hipDeviceGetDefaultMemPool(&defaultPool, 0);

    // Show pool usage before allocation
    printPoolUsage(defaultPool, "Before Allocation");

    // Allocate device memory using default memory pool
    hipMallocAsync((void**)&d_A, size, stream);
    hipMallocAsync((void**)&d_B, size, stream);
    hipMallocAsync((void**)&d_C, size, stream);

    // Wait for allocation to complete
    hipStreamSynchronize(stream);

    // Show pool usage after allocation
    printPoolUsage(defaultPool, "After Allocation");

    // Asynchronous memory copy to device
    hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, stream);

    // Launch kernel
    vectorAdd<<<(N + 255) / 256, 256, 0, stream>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpyAsync(h_C, d_C, size, hipMemcpyDeviceToHost, stream);

    // Wait for all operations to finish
    hipStreamSynchronize(stream);

    printf("C[100] = %f\n", h_C[100]);  // Should be 100 + 2*100 = 300

    // Free device memory asynchronously
    hipFreeAsync(d_A, stream);
    hipFreeAsync(d_B, stream);
    hipFreeAsync(d_C, stream);

    // Final sync before cleanup
    hipStreamSynchronize(stream);

    // Show pool usage after deallocation (memory is returned to the pool, not system)
    printPoolUsage(defaultPool, "After Free (Pool still holds memory)");

    free(h_A);
    free(h_B);
    free(h_C);
    hipStreamDestroy(stream);

    return 0;
}
