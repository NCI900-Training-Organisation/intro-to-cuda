#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel: simple vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int N) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

void printPoolUsage(hipMemPool_t pool, const char* label) {
    size_t current, high;
    hipMemPoolGetAttribute(pool, hipMemPoolAttrReservedMemCurrent, &current);
    hipMemPoolGetAttribute(pool, hipMemPoolAttrReservedMemHigh, &high);

    printf("%s:\n", label);
    printf("  Reserved memory:      %lu bytes (%.2f MB)\n", current, current / (1024.0 * 1024));
    printf("  Peak reserved memory: %lu bytes (%.2f MB)\n\n", high, high / (1024.0 * 1024));
}

int main() 
{
    const int N = 1 << 16;
    const int size = N * sizeof(float);

    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize host input arrays
    for (int i = 0; i < N; ++i) {
        h_A[i] = i;
        h_B[i] = 2 * i;
    }

    float *d_A, *d_B, *d_C;
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Create a custom memory pool for device 0
    hipMemPool_t myPool;
    hipMemPoolProps props = {};
    props.allocType = hipMemAllocationTypePinned; // Pinned host memory
    props.handleTypes = hipMemHandleTypeNone;     // No interprocess sharing
    props.location.type = hipMemLocationTypeDevice;
    props.location.id = 0;                          // GPU device 0

    hipMemPoolCreate(&myPool, &props);

    // Configure the custom memory pool attributes
    int threshold = 1024 * 1024; // 1 MB threshold for releasing memory
    hipMemPoolSetAttribute(myPool, hipMemPoolAttrReleaseThreshold, &threshold);          // 1 MB threshold
    int current = 512 * 1024 * 1024;
    hipMemPoolSetAttribute(myPool, hipMemPoolAttrReservedMemCurrent, &current);  // 512 MB reserved (informational)
    int high = 1024 * 1024 * 1024; // 1 GB high limit
    hipMemPoolSetAttribute(myPool, hipMemPoolAttrReservedMemHigh, &high);    // 1 GB high limit (informational)
    //cudaMemPoolSetAttribute(myPool, cudaMemPoolAttrReusePolicy, cudaMemPoolReusePolicyAggressive);  // Aggressive reuse
    //cudaMemPoolSetAttribute(myPool, cudaMemPoolAttrAllocationGranularity, 256 * 1024);      // 256 KB granularity

    // Show pool usage before allocation
    printPoolUsage(myPool, "Before Allocation");

    // Allocate device memory using the custom memory pool asynchronously
    hipMallocFromPoolAsync((void**)&d_A, size, myPool, stream);
    hipMallocFromPoolAsync((void**)&d_B, size, myPool, stream);
    hipMallocFromPoolAsync((void**)&d_C, size, myPool, stream);

    // Wait for allocations to complete
    hipStreamSynchronize(stream);

    // Show pool usage after allocation
    printPoolUsage(myPool, "After Allocation");

    // Copy data from host to device asynchronously
    hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, stream);

    // Launch kernel
    vectorAdd<<<(N + 255) / 256, 256, 0, stream>>>(d_A, d_B, d_C, N);

    // Copy result back to host asynchronously
    hipMemcpyAsync(h_C, d_C, size, hipMemcpyDeviceToHost, stream);

    // Wait for all operations to finish
    hipStreamSynchronize(stream);

    printf("C[100] = %f\n", h_C[100]);  // Should be 100 + 2*100 = 300

    // Free device memory asynchronously
    hipFreeAsync(d_A, stream);
    hipFreeAsync(d_B, stream);
    hipFreeAsync(d_C, stream);

    // Synchronize before trimming
    hipStreamSynchronize(stream);

    // Trim the pool to release unused memory back to the system
    hipMemPoolTrimTo(myPool, 0);  // Release all unused memory

    // Show pool usage after freeing and trimming
    printPoolUsage(myPool, "After Free and Trim");

    // Cleanup
    free(h_A);
    free(h_B);
    free(h_C);
    hipStreamDestroy(stream);

    // Destroy the custom memory pool
    hipMemPoolDestroy(myPool);

    return 0;
}
