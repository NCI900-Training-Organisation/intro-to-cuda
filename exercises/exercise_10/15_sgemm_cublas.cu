#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define IDX2C(i,j,ld) (((j)*(ld))+(i))  // Column-major index

void matrixMultiplyCuBLAS() 
{
    const int N = 2;
    const float alpha = 1.0f;
    const float beta = 0.0f;

    float h_A[N * N] = {1, 2, 3, 4};   // Row-major layout
    float h_B[N * N] = {1, 2, 3, 4};
    float h_C[N * N] = {0, 0, 0, 0};

    float *d_A, *d_B, *d_C;

    hipMalloc((void**)&d_A, N * N * sizeof(float));
    hipMalloc((void**)&d_B, N * N * sizeof(float));
    hipMalloc((void**)&d_C, N * N * sizeof(float));

    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // C = A * B (row-major inputs treated as column-major)
    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,  //TOD0: change to CUBLAS_OP_N
                N, N, N,
                &alpha,
                d_A, N,
                d_B, N,
                &beta,
                d_C, N);

    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Result C = A x B:\n");
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%.1f ", h_C[IDX2C(i, j, N)]);
        }
        printf("\n");
    }

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    matrixMultiplyCuBLAS();
    return 0;
}
