#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float *A, const float *B, float *C, int N) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

void checkCuda(hipError_t err, const char* msg) 
{
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(1);
    }
}

int main() 
{
    const int N = 1 << 20;
    const int size = N * sizeof(float);
    const int iterations = 100;

    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    for (int i = 0; i < N; ++i) {
        h_A[i] = i;
        h_B[i] = 2 * i;
    }

    float *d_A, *d_B, *d_C;
    checkCuda(hipMalloc(&d_A, size), "Alloc d_A");
    checkCuda(hipMalloc(&d_B, size), "Alloc d_B");
    checkCuda(hipMalloc(&d_C, size), "Alloc d_C");

    checkCuda(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice), "Memcpy h_A");
    checkCuda(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice), "Memcpy h_B");

    hipEvent_t start, stop;
    hipStream_t stream;
    hipStreamCreate(&stream);
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // --- Traditional execution ---
    checkCuda(hipEventRecord(start, stream), "Start traditional");
    for (int i = 0; i < iterations; ++i) {
        vectorAdd<<<(N + 255)/256, 256, 0, stream>>>(d_A, d_B, d_C, N);
        hipMemcpyAsync(h_C, d_C, size, hipMemcpyDeviceToHost, stream);
    }
    checkCuda(hipEventRecord(stop, stream), "Stop traditional");
    hipEventSynchronize(stop);
    float time_traditional = 0.0f;
    hipEventElapsedTime(&time_traditional, start, stop);
    printf("[Traditional] Time: %.3f ms (avg %.3f ms/iteration)\n",
           time_traditional, time_traditional / iterations);

    // --- CUDA Graph execution ---
    hipGraph_t graph;
    hipGraphExec_t graphExec;

    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    vectorAdd<<<(N + 255)/256, 256, 0, stream>>>(d_A, d_B, d_C, N);
    hipMemcpyAsync(h_C, d_C, size, hipMemcpyDeviceToHost, stream);
    hipStreamEndCapture(stream, &graph);
    hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);

    checkCuda(hipEventRecord(start, stream), "Start graph");
    for (int i = 0; i < iterations; ++i) {
        hipGraphLaunch(graphExec, stream);
    }
    checkCuda(hipEventRecord(stop, stream), "Stop graph");
    hipEventSynchronize(stop);
    float time_graph = 0.0f;
    hipEventElapsedTime(&time_graph, start, stop);
    printf("[Graph]       Time: %.3f ms (avg %.3f ms/iteration)\n",
           time_graph, time_graph / iterations);

    // --- Cleanup ---
    hipGraphExecDestroy(graphExec);
    hipGraphDestroy(graph);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipStreamDestroy(stream);

    return 0;
}
