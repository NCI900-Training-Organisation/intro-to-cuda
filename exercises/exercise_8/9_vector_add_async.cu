#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel: simple vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int N) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

int main() 
{
    const int N = 1 << 16;
    const int size = N * sizeof(float);

    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize host input arrays
    for (int i = 0; i < N; ++i) {
        h_A[i] = i;
        h_B[i] = 2 * i;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipStream_t stream;
    hipStreamCreate(&stream);

    // Begin CUDA Graph capture
    hipGraph_t graph;
    hipGraphExec_t graphExec;
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

    // Copy inputs host->device (inside capture)
    hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, stream);

    // Kernel launch (recorded)
    vectorAdd<<<(N + 255)/256, 256, 0, stream>>>(d_A, d_B, d_C, N);

    // Copy result to host (recorded)
    hipMemcpyAsync(h_C, d_C, size, hipMemcpyDeviceToHost, stream);

    // End capture
    hipStreamEndCapture(stream, &graph);

    // Instantiate executable graph
    hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);

    // Execute the graph multiple times
    for (int i = 0; i < 5; ++i) {
        hipGraphLaunch(graphExec, stream);
        hipStreamSynchronize(stream);
        printf("Run %d: C[100] = %f\n", i, h_C[100]);  // Should be 300 (100 + 200)
    }

    // Clean up
    hipGraphExecDestroy(graphExec);
    hipGraphDestroy(graph);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    hipStreamDestroy(stream);

    return 0;
}
