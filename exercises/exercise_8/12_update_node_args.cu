#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void square(float *d_data, int N) 
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N)
        d_data[idx] = d_data[idx] * d_data[idx];
}

void check(hipError_t err, const char* msg) 
{
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main() 
{
    const int N = 1 << 16;
    const int size = N * sizeof(float);

    // Host buffers
    float *h_input[3], *h_output[3];
    for (int i = 0; i < 3; ++i) {
        h_input[i] = (float*)malloc(size);
        h_output[i] = (float*)malloc(size);
        for (int j = 0; j < N; ++j)
            h_input[i][j] = (float)(j + i * 1000);  // Different input data
    }

    // Device buffers
    float *d_data[3];
    for (int i = 0; i < 3; ++i) {
        check(hipMalloc(&d_data[i], size), "Alloc d_data");
    }

    hipStream_t stream;
    check(hipStreamCreate(&stream), "Create stream");

    // Create graph
    hipGraph_t graph;
    check(hipGraphCreate(&graph, 0), "Create graph");

    // Memset node for d_data[0] initially (just to fill with zero)
    hipMemsetParams memsetParams = {};
    memsetParams.dst = d_data[0];
    memsetParams.value = 0;
    memsetParams.pitch = 0;
    memsetParams.elementSize = sizeof(float);
    memsetParams.width = N;
    memsetParams.height = 1;
    hipGraphNode_t memsetNode;
    check(hipGraphAddMemsetNode(&memsetNode, graph, NULL, 0, &memsetParams), "Add memset");

    // Memcpy H2D node for d_data[0]
    hipMemcpy3DParms copyH2D = {};
    copyH2D.srcPtr = make_hipPitchedPtr(h_input[0], size, N, 1);
    copyH2D.dstPtr = make_hipPitchedPtr(d_data[0], size, N, 1);
    copyH2D.extent = make_hipExtent(size, 1, 1);
    copyH2D.kind = hipMemcpyHostToDevice;
    hipGraphNode_t memcpyH2DNode;
    check(hipGraphAddMemcpyNode(&memcpyH2DNode, graph, NULL, 0, &copyH2D), "Add memcpy H2D");

    // Kernel node — dummy args for now, will update dynamically
    int n = N;
    void* kernelArgs[] = { &d_data[0], &n };
    hipKernelNodeParams kernelParams = {};
    kernelParams.func = (void*)square;
    kernelParams.gridDim = dim3((N + 255) / 256);
    kernelParams.blockDim = dim3(256);
    kernelParams.kernelParams = kernelArgs;
    kernelParams.extra = NULL;
    hipGraphNode_t kernelNode;
    check(hipGraphAddKernelNode(&kernelNode, graph, NULL, 0, &kernelParams), "Add kernel");

    // Memcpy D2H node for d_data[0]
    hipMemcpy3DParms copyD2H = {};
    copyD2H.srcPtr = make_hipPitchedPtr(d_data[0], size, N, 1);
    copyD2H.dstPtr = make_hipPitchedPtr(h_output[0], size, N, 1);
    copyD2H.extent = make_hipExtent(size, 1, 1);
    copyD2H.kind = hipMemcpyDeviceToHost;
    hipGraphNode_t memcpyD2HNode;
    check(hipGraphAddMemcpyNode(&memcpyD2HNode, graph, NULL, 0, &copyD2H), "Add memcpy D2H");

    // Dependencies: memset -> memcpyH2D -> kernel -> memcpyD2H
    check(hipGraphAddDependencies(graph, &memsetNode, &memcpyH2DNode, 1), "memset -> memcpyH2D");
    check(hipGraphAddDependencies(graph, &memcpyH2DNode, &kernelNode, 1), "memcpyH2D -> kernel");
    check(hipGraphAddDependencies(graph, &kernelNode, &memcpyD2HNode, 1), "kernel -> memcpyD2H");

    // Instantiate graph exec
    hipGraphExec_t graphExec;
    check(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0), "Instantiate graph");

    // Now loop for all 3 data sets, update kernel and memcpy nodes dynamically and launch
    for (int i = 0; i < 3; ++i) {
        // Update memcpy H2D node params
        copyH2D.srcPtr = make_hipPitchedPtr(h_input[i], size, N, 1);
        copyH2D.dstPtr = make_hipPitchedPtr(d_data[i], size, N, 1);
        check(hipGraphExecMemcpyNodeSetParams(graphExec, memcpyH2DNode, &copyH2D), "Update memcpy H2D params");

        // Update kernel node params
        int n = N;
        void* kernelArgsNew[] = { &d_data[i], &n };
        hipKernelNodeParams kernelParamsNew = {};
        kernelParamsNew.func = (void*)square;
        kernelParamsNew.gridDim = dim3((N + 255) / 256);
        kernelParamsNew.blockDim = dim3(256);
        kernelParamsNew.kernelParams = kernelArgsNew;
        kernelParamsNew.extra = NULL;
        check(hipGraphExecKernelNodeSetParams(graphExec, kernelNode, &kernelParamsNew), "Update kernel params");

        // Update memcpy D2H node params
        copyD2H.srcPtr = make_hipPitchedPtr(d_data[i], size, N, 1);
        copyD2H.dstPtr = make_hipPitchedPtr(h_output[i], size, N, 1);
        check(hipGraphExecMemcpyNodeSetParams(graphExec, memcpyD2HNode, &copyD2H), "Update memcpy D2H params");

        // Launch graph
        check(hipGraphLaunch(graphExec, stream), "Launch graph");
        check(hipStreamSynchronize(stream), "Sync stream");

        // Check results
        printf("Result sample for buffer %d: h_output[%d] = %f\n", i, 10, h_output[i][10]);
    }

    // Cleanup
    hipGraphExecDestroy(graphExec);
    hipGraphDestroy(graph);
    hipStreamDestroy(stream);
    for (int i = 0; i < 3; ++i) {
        hipFree(d_data[i]);
        free(h_input[i]);
        free(h_output[i]);
    }

    return 0;
}
