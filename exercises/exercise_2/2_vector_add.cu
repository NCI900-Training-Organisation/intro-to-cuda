#include <stdio.h>
#include <hip/hip_runtime.h> // Provides access to CUDA runtime API functions


__global__ void add_vectors(float *a, float *b, float *c, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n)
        c[idx] = a[idx] + b[idx];
}

int main()
{
    int n = 1024;
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;

    // Allocate host memory
    h_a = (float*)malloc(n * sizeof(float));
    h_b = (float*)malloc(n * sizeof(float));
    h_c = (float*)malloc(n * sizeof(float));

    // Initialize host arrays
    for (int i = 0; i < n; i++) {
        h_a[i] = i;
        h_b[i] = i;
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, n * sizeof(float));
    hipMalloc((void**)&d_b, n * sizeof(float));
    hipMalloc((void**)&d_c, n * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel with 256 threads per block and enough blocks to cover all elements
    add_vectors<<<(n + 255) / 256, 256>>>(d_a, d_b, d_c, n);

    // Copy result back to host
    hipMemcpy(h_c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}